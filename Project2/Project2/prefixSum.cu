#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "prefixSum.h"
#include <cmath>
#include <thrust/random.h>
float a, b ,c;
#define blocksize  128
void checkCUDAError(const char *msg, int line)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        if( line >= 0 )
        {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
        exit(EXIT_FAILURE); 
    }
} 

__global__ void scan(float *arr, float *result, int n){
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if(index < n){
		/*for(int offset = 1; offset < n-1; offset*2){
			if(index >= offset){
				result[index] = arr[index - offset] + arr[index];
			}
			else {
				result[index] = arr[index];
			}
			__syncthreads();

			float *temp = arr;
			arr = result;
			result = temp;
		}*/
		int logn = ceil(log(float(n))/log(2.0f));
		for (int d=1; d<=logn; d++){    
			int offset = powf(2.0f, d-1); 
			if (index >= offset){
				result[index] = arr[index-offset] + arr[index];
			}
			else{
				result[index] = arr[index]; 
			}
			__syncthreads();

			float* temp = arr;
			arr = result;
			result = temp;
		  }
	}
}


__global__ void scanSharedSingleBlock(float *arr, float *result, int n){
	extern __shared__ float temp[];
	int index = threadIdx.x;
	if(index < n){
		int pout = 0, pin = 1;
		temp[pout*n + index] = arr[index];
		__syncthreads();
		for(int offset = 1; offset < n; offset *= 2){
			pout = 1 - pout;
			pin =  1- pout;
			if(index >= offset)
				temp[pout*n + index] = temp[pin*n + index - offset] + temp[pin*n +index];
			else
				temp[pout*n + index] = temp[pin*n + index];
			__syncthreads();
		}
		result[index] = temp[pout*n + index];
	}
}

__global__ void scanSharedArbitraryLength(float *arr, float *result, int n, float* sums){
	extern __shared__ float temp[];
	
	int index = threadIdx.x;
	int globalIndex = threadIdx.x + (blockIdx.x * blockDim.x);
	if(globalIndex < n){
		float *tempIn = &temp[0];
		float *tempOut = &temp[n];
	
		tempOut[index] = arr[globalIndex];
		__syncthreads();
		for(int offset = 1; offset < n; offset *= 2){
			float* temp = tempIn;
			tempIn = tempOut;
			tempOut = temp;
			//__syncthreads();
			if(index >= offset){
				tempOut[index] = tempIn[index - offset] + tempIn[index];
			}
			else{
				tempOut[index] = tempIn[index];
			}
			
			__syncthreads();
		}
		result[globalIndex] = tempOut[index];
		if(index == blocksize -1) 
			sums[blockIdx.x] = tempOut[index];//last element in this block
	}
}
__global__ void getIncr(float* arr, float* result, int n, int d){
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if(index < n){
		if(index >= (int)pow(2.0,d-1)){
			result[index] = arr[index - d] + arr[index];
		}
		else{
			result[index] = arr[index];
		}
		
	}
}
__global__ void addIncr(float *Incr, float *arr, int n){
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if(index < n){
		if(blockIdx.x >= 1){
			arr[index] += Incr[blockIdx.x-1];
		}
	}
}
__global__ void scatterSetup(float *arr, float *result, int n){
	__shared__ float temp[blocksize];
	__shared__ float temp2[blocksize];
	int globalIndex = threadIdx.x + (blockIdx.x * blockDim.x);
	int index = threadIdx.x;
	temp[index] = arr[globalIndex];
	__syncthreads();
	if(globalIndex < n){
		if(temp[index] == 0)
			temp2[index] = 0;
		else 
			temp2[index] = 1;
		__syncthreads();

		for(int offset = 1; offset <= blocksize; offset*=2){
			if(index >= offset){
				temp[index] = temp2[index - offset] + temp2[index];
			}
			else{
				temp[index] = temp2[index];
			}
			temp2[index] = temp[index];
			__syncthreads();
		}
		result[globalIndex] = temp2[index];
	}
}
__global__ void ScanAdd (float *arr, float *b, int size){
	__shared__ int temp[blocksize];

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	temp[threadIdx.x] = arr[index];
	__syncthreads();

	for(int b = 0; b < blockIdx.x ; ++b){
		temp[threadIdx.x] += arr[ (b + 1) * blocksize - 1];
	}
	b[index] = temp[threadIdx.x];
}
__global__ void scatterShift(float *arr, float *result, int n){
	
	//__shared__ int temp[blocksize];
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if(index == 0){		
		result[index] = 0;
	}
	else 
		result[index] = arr[index - 1];
	
}
__global__ void scatter(float *arr, float *arr_scan, float *result, int n){
	__shared__ float temp[blocksize];
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	temp[threadIdx.x] = arr[index];
	__syncthreads();

	if(temp[threadIdx.x] != 0){
		int newindex = (int)arr_scan[index];
		result[newindex] = temp[threadIdx.x];
	}
}
int padn(int n){
	int i;
	for(i = 1; n > i*blocksize; i++){}
	return i*blocksize;
}
void shiftRight(float * arr, int n){
	for(int i = n-1; i > 0; i--){
		arr[i] = arr[i-1];	
	}
	arr[0] = 0;
}
void scanGPU(int n, float *in_arr, float *out_arr, int type){

	//dim3 dimBlock(1, 1);//how to decide?
	//dim3 dimGrid(n, 1); 
	
	dim3 fullBlocksPerGrid((int)ceil(float(n)/float(blocksize)));
	dim3 threadsPerBlock(blocksize); 

	int size =n*sizeof(float);
	float *in_arr_d, *out_arr_d;
	hipMalloc((void**)&in_arr_d, size);
	hipMemcpy(in_arr_d, in_arr, size, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("Kernel failed!");
	hipMalloc((void**)&out_arr_d, size);
	checkCUDAErrorWithLine("Kernel failed!");
	int n_round = padn(n);
	//-----------------naive global-----------------------------------------
	if(type == 1){
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord( start, 0);
		for(int i = 0; i < itertimes; i++)
			scan<<<fullBlocksPerGrid, threadsPerBlock>>>(in_arr_d, out_arr_d, n_round);
		hipEventRecord( stop, 0);
		hipEventSynchronize( stop );
		float time = 0.0f;
		hipEventElapsedTime( &time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);

		printf(" %.4f ms \n", time);
	}
	//----------------shared single block-----------------------------------
	if(type == 2){
		
		

			int sumNum = (int)ceil(float(n)/float(blocksize));
			if(sumNum < 1){
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord( start, 0);
				for(int i = 0; i < itertimes; i++){
					scanSharedSingleBlock<<<fullBlocksPerGrid, threadsPerBlock, 2*n*sizeof(float)>>>(in_arr_d, out_arr_d, n);
				}
				hipEventRecord( stop, 0);
				hipEventSynchronize( stop );
				float time = 0.0f;
				hipEventElapsedTime( &time, start, stop);
				hipEventDestroy(start);
				hipEventDestroy(stop);

				printf(" %.4f ms \n", time);
				checkCUDAErrorWithLine("Kernel failed!");
			}
		//----------------shared arbitrary length-------------------------------
			else{
				float *sums_d, *incr_d;
				int sumsize = sumNum * sizeof(float);
				hipMalloc((void**)&sums_d, sumsize);
				hipMalloc((void**)&incr_d, sumsize);
				
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord( start, 0);
				for(int i = 0; i < itertimes; i++){
					scanSharedArbitraryLength<<<fullBlocksPerGrid, threadsPerBlock, 2*n_round*sizeof(float)>>>(in_arr_d, out_arr_d, n_round, sums_d);
				}
				hipEventRecord( stop, 0);
				hipEventSynchronize( stop );
				float time = 0.0f;
				hipEventElapsedTime( &time, start, stop);
				hipEventDestroy(start);
				hipEventDestroy(stop);

				printf(" %.4f ms \n", time);
				checkCUDAErrorWithLine("Kernel failed!");
				int sumNum_round = padn(sumNum);
				scan<<<fullBlocksPerGrid, threadsPerBlock>>>(sums_d, incr_d, sumNum_round);

				/*dim3 sumBlocksPerGrid((int)ceil(sumNum/(float)blocksize));
				for(int d = 1; (int)pow(2.0,d-1) <= sumNum ;d++){
					getIncr<<<sumBlocksPerGrid, threadsPerBlock>>>(sums_d, incr_d, sumNum, d);
					hipDeviceSynchronize();
					float *temp = sums_d;
					sums_d = incr_d;
					incr_d = temp;
				}*/
				checkCUDAErrorWithLine("Kernel failed!");
				addIncr<<<fullBlocksPerGrid, threadsPerBlock>>>(incr_d, out_arr_d, n);
				checkCUDAErrorWithLine("Kernel failed!");
				hipDeviceSynchronize();
				hipFree(sums_d);
				hipFree(incr_d);
			}
		
		
	}
	

	
	//----------------copy to host and shift------------------------------------
	hipMemcpy(out_arr, out_arr_d, size, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("Kernel failed!");
	shiftRight(out_arr, n);
	checkCUDAErrorWithLine("Kernel failed!");

	hipFree(in_arr_d);
	hipFree(out_arr_d);
	
}

void scatterGPU(int n, float *in_arr, float *out_arr){
	dim3 fullBlocksPerGrid((int)ceil(float(n)/float(blocksize)));
	dim3 threadsPerBlock(blocksize); 

	int size =n*sizeof(float);
	float *in_arr_d, *out_arr_d, *arr_preScan, *arr_postScan, *arr_scan;
	hipMalloc((void**)&in_arr_d, size);
	hipMemcpy(in_arr_d, in_arr, size, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("Kernel failed!");
	hipMalloc((void**)&out_arr_d, size);
	checkCUDAErrorWithLine("Kernel failed!");
	hipMalloc((void**)&arr_preScan, size);
	hipMalloc((void**)&arr_postScan, size);
	hipMalloc((void**)&arr_scan, size);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0);
	for(int i = 0; i < itertimes; i++){
		scatterSetup<<<fullBlocksPerGrid, threadsPerBlock>>>(in_arr_d, arr_preScan, n);
		ScanAdd<<<fullBlocksPerGrid, threadsPerBlock>>>(arr_preScan, arr_postScan, n);
		scatterShift<<<fullBlocksPerGrid, threadsPerBlock>>>(arr_postScan, arr_scan, n);
		scatter<<<fullBlocksPerGrid, threadsPerBlock>>>(in_arr_d, arr_scan, out_arr_d, n);
	}
	hipEventRecord( stop, 0);
	hipEventSynchronize( stop );
	float time = 0.0f;
	hipEventElapsedTime( &time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf(" %.4f ms \n", time);
	
	checkCUDAErrorWithLine("Kernel failed!");
	//hipDeviceSynchronize();
	hipMemcpy(out_arr, out_arr_d, size, hipMemcpyDeviceToHost);

	hipFree(in_arr_d);
	hipFree(out_arr_d);
	hipFree(arr_preScan);
	hipFree(arr_postScan);
	hipFree(arr_scan);

}
